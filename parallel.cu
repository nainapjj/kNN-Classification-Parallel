#include <stdio.h>
#include <algorithm>
#include <cmath>
#include <iostream>
#include <fstream>
#include <ctime>
#include <string>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define MAX 10
#define MIN 0

#define FILE_NAME "small12345.txt"
#define K 2

using namespace std;

__global__ void normalize(double * d_input, double *d_max, double *d_min, unsigned int numAttributes, 
    unsigned int numElems) {
    
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int attributeIdx = tid % numAttributes;
    
    if(tid < numElems*numAttributes) {
        d_input[tid] = (d_input[tid] - d_min[attributeIdx]) / (d_max[attributeIdx] - d_min[attributeIdx]);
    }
}

__global__ void findDistance(double *d_inputNormal, double *d_inputSample,  double *d_output, unsigned int numElems) {
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < numElems) {
        d_output[tid] = (d_inputNormal[tid] - d_inputSample[tid])*(d_inputNormal[tid] - d_inputSample[tid]);
    }

}

/*__global__ void block_sum(double *input, double *results, size_t n)
{
    extern __shared__ float sdata[];
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int tx = threadIdx.x;
    // load input into __shared__ memory
    float x = 0;
    if(i < n) {
        x = input[i];
    }
    sdata[tx] = x; 
    
    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
      if(tx < offset)
      {
        // add a partial sum upstream to our own
        sdata[tx] += sdata[tx + offset];
      }
      __syncthreads();
    } 
    
    if(tx == 0) {
        results[blockIdx.x] = 
    }
}*/

void parse(int* numAttributes, int* numKnownSamples, int* numClass, int *numUnknowns,
    double ** min, double ** max, double ** knowns, int ** classifications, 
    double ** unknowns, string** unknownNames)
{
    ifstream myfile(FILE_NAME, ios::in);  // declare and open
    
    int numAttrib, numKnownSamp, numCla, numUn;
    myfile >> numKnownSamp >> numAttrib >> numCla >> numUn;
    
    *numAttributes = numAttrib;
    *numKnownSamples = numKnownSamp;
    *numClass = numCla;
    *numUnknowns = numUn;
    
    // Populate all of the mins and maxes
    *min = (double*) malloc(sizeof(double) * numAttrib);
    *max = (double*) malloc(sizeof(double) * numAttrib);
    for (int i = 0; i < numAttrib; i++) {
        int currentMax, currentMin;
        myfile >> currentMin >> currentMax;
        (*min)[i] = currentMin;
        (*max)[i] = currentMax;
    }
    
    
    // Populate the known object types
    *classifications =(int*) malloc(sizeof(int) * numKnownSamp);
    *knowns = (double*) malloc(sizeof(double) * numKnownSamp * numAttrib);
    
    for (int i = 0; i < numKnownSamp; i++) {
        int currentClass;
        myfile >> currentClass;
        (*classifications)[i] = currentClass;
        
        for (int j = 0; j < numAttrib; j++) {
            double currentAttrib;
            myfile >> currentAttrib;
            (*knowns)[i*numAttrib + j] = currentAttrib;
        }
    }
    
    // Populate the unknown object types
    *unknownNames = new string[numUn];
    *unknowns = (double*) malloc(sizeof(double) * numUn * numAttrib);
    
    for (int i = 0; i < numUn; i++) {
        string currentName;
        myfile >> currentName;
        (*unknownNames)[i] = currentName;
        
        for (int j = 0; j < numAttrib; j++) {
            double currentAttrib;
            myfile >> currentAttrib;
            (*unknowns)[i*numAttrib + j] = currentAttrib;
        }
    }
    
    myfile.close();
}



//this is pseudocode
int main() {
    unsigned int numBlocks = 512;
    unsigned int threadsPerBlock = 256;
    
    // Metadata about our learning algorithm data
    int numAttributes, numKnownSamples, numClass, numUnknowns;
    
    // Data that needs to be sent to the device.
    double *h_min, *h_max;
    double *h_knowns;
    int *h_classifications;
    double *h_unknowns;
    
    // Device data
    double *d_min, *d_max;
    double *d_knowns;
    int *d_classifications;
    double *d_unknowns;
    
    string *unknownNames;
    
    // Needed for the profiling
    std::clock_t start;
    double duration;
    
    parse(&numAttributes, &numKnownSamples, &numClass, &numUnknowns, 
        &h_min, &h_max, &h_knowns, &h_classifications, &h_unknowns, &unknownNames);
    
    // Start mallocing the data to the kernel
    hipMalloc(&d_min, sizeof(double) * numAttributes);
    hipMalloc(&d_max, sizeof(double) * numAttributes);
    hipMalloc(&d_knowns, sizeof(double) * numKnownSamples * numAttributes);
    hipMalloc(&d_unknowns, sizeof(double) * numUnknowns * numAttributes);
    hipMalloc(&d_classifications, sizeof(int) * numKnownSamples);
    
    // Copy the data from the host to the kernel
    hipMemcpy(d_min, h_min, sizeof(double) * numAttributes, hipMemcpyHostToDevice);
    hipMemcpy(d_max, h_max, sizeof(double) * numAttributes, hipMemcpyHostToDevice);
    hipMemcpy(d_knowns, h_knowns, sizeof(double) * numKnownSamples * numAttributes, hipMemcpyHostToDevice);
    hipMemcpy(d_unknowns, h_unknowns, sizeof(double) * numUnknowns * numAttributes, hipMemcpyHostToDevice);
    hipMemcpy(d_classifications, h_classifications, sizeof(int) * numKnownSamples, hipMemcpyHostToDevice);

    
    // Normalize the known values
    threadsPerBlock = 256;
    numBlocks = numAttributes * numKnownSamples / threadsPerBlock;
    normalize<<<numBlocks, threadsPerBlock>>>(d_knowns, d_max, d_min, 
        numAttributes, numKnownSamples);
    
    // Normalize the unknown values
    threadsPerBlock = 256;
    numBlocks = numAttributes * numKnownSamples / threadsPerBlock;
    normalize<<<numBlocks, threadsPerBlock>>>(d_unknowns, d_max, d_min, 
        numAttributes, numUnknowns);
    
    hipMemcpy(h_unknowns, d_unknowns, sizeof(double) * numUnknowns * numAttributes, hipMemcpyDeviceToDevice);
    
    for (int i = 0; i < 5; i++) {
        printf("%d ", h_unknowns[i]); 
    }
    printf("\n");



    //findDistance<<<numBlocks, threadsPerBlock>>>(d_unknowns, 
}
