#include <stdio.h>
#include <algorithm>
#include <cmath>
#include <iostream>
#include <fstream>
#include <ctime>
#include <string>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define MAX 10
#define MIN 0

#define FILE_NAME "input.txt"
#define K 2

using namespace std;

__global__ void normalize(float * d_input, float *d_max, float *d_min, unsigned int numAttributes, 
    unsigned int numElems) {
    
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int attributeIdx = tid % numAttributes;
    
    if(tid < numElems*numAttributes) {
        d_input[tid] = (d_input[tid] - d_min[attributeIdx]) / (d_max[attributeIdx] - d_min[attributeIdx]);
    }
}

__global__ void findDistanceV2(float *d_inputAttributes, float **d_inputSample,  float *d_output, unsigned int numAttributes, 
    unsigned int numSamples) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(row < numSamples && col < numAttributes) {
        d_output[col+row*numAttributes] = (d_inputAttributes[col] - d_inputSample[row][col])*(d_inputAttributes[col] - d_inputSample[row][col]);
    }


}

__global__ void findDistance(float *d_inputAttributes, float *d_inputSample,  float *d_output, unsigned int numAttributes, 
    unsigned int numElems) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    float distance = 0;
    
    if (tid < numElems) {
        for (int i = 0; i < numAttributes; i++) {
            distance += pow(d_inputAttributes[numAttributes*tid + i] - d_inputSample[i], 2);
        }
        
        // OPTIMIZATION: We don't have to square root, because if 
        // there's no point in wasting all of the distance values are squares
        d_output[tid] = distance;
    }
}

//  unsigned int *d_val;
//  unsigned int *d_pos;
//	unsigned int exp = (unsigned int)log2((float)numElems)+1;
//	unsigned int padding = (unsigned int)exp2((float)exp);
//for (int i = 2; i<=padding; i=i*2)
//    {
//		 for (int inr = i/2; inr>=1; inr=inr/2)
//		{
//			//use bitonic sort
//			bitonic_sort<<<grid,block>>> (d_val, d_pos, padding, i, inr);
//		}
//	}


__global__ void bitonic_sort (unsigned int*  d_val, unsigned int*  d_pos, 
							  const int  padding, const int  count, const int inr)
{
  
		//bitonic_sort_ <S> <<<gsize,bsize>>>(d_val, d_pos, padding, count, inr);
		unsigned int up, down;
		unsigned int up_, down_;
	    unsigned int id = threadIdx.x+threadIdx.y*blockDim.x+blockIdx.x*blockDim.x*blockDim.y;
		int updown = 0;//, inc, inr;
		int pass=0;

		if (id <padding)
		{
			updown = (id/count) % 2;
			//determines the direction of the comparison 
			up = d_val[id];
			up_ = d_pos[id];
			if (id % (inr*2) < inr)
			{
				down = d_val[id+inr];
				down_= d_pos[id+inr];
				//The output is a sorted list that is ascending if up is true
				pass = ((int)(up>=down)==updown);

				if (!pass)  
				{
					d_val[id]=down;
					d_pos[id]=down_;
					d_val[id+inr]=up;
					d_pos[id+inr]=up_;
				}
			}
			
		}
		return;
    
}

__global__ void bitonicSort(float *d_distance, int numAttributes) 
{
    
}

/*__global__ void block_sum(float *input, float *results, size_t n)
{
    extern __shared__ float sdata[];
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    int tx = threadIdx.x;
    // load input into __shared__ memory
    float x = 0;
    if(i < n) {
        x = input[i];
    }
    sdata[tx] = x; 
    
    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
      if(tx < offset)
      {
        // add a partial sum upstream to our own
        sdata[tx] += sdata[tx + offset];
      }
      __syncthreads();
    } 
    
    if(tx == 0) {
        results[blockIdx.x] = 
    }
}*/

void parse(int* numAttributes, int* numKnownSamples, int* numClass, int *numUnknowns,
    float ** min, float ** max, float ** knowns, int ** classifications, 
    float ** unknowns, string** unknownNames)
{
    ifstream myfile(FILE_NAME, ios::in);  // declare and open
    
    int numAttrib, numKnownSamp, numCla, numUn;
    myfile >> numKnownSamp >> numAttrib >> numCla >> numUn;
    
    *numAttributes = numAttrib;
    *numKnownSamples = numKnownSamp;
    *numClass = numCla;
    *numUnknowns = numUn;
    
    // Populate all of the mins and maxes
    *min = (float*) malloc(sizeof(float) * numAttrib);
    *max = (float*) malloc(sizeof(float) * numAttrib);
    for (int i = 0; i < numAttrib; i++) {
        int currentMax, currentMin;
        myfile >> currentMin >> currentMax;
        (*min)[i] = currentMin;
        (*max)[i] = currentMax;
    }
    
    
    // Populate the known object types
    *classifications =(int*) malloc(sizeof(int) * numKnownSamp);
    *knowns = (float*) malloc(sizeof(float) * numKnownSamp * numAttrib);
    
    for (int i = 0; i < numKnownSamp; i++) {
        int currentClass;
        myfile >> currentClass;
        (*classifications)[i] = currentClass;
        
        for (int j = 0; j < numAttrib; j++) {
            float currentAttrib;
            myfile >> currentAttrib;
            (*knowns)[i*numAttrib + j] = currentAttrib;
        }
    }
    
    // Populate the unknown object types
    *unknownNames = new string[numUn];
    *unknowns = (float*) malloc(sizeof(float) * numUn * numAttrib);
    
    for (int i = 0; i < numUn; i++) {
        string currentName;
        myfile >> currentName;
        (*unknownNames)[i] = currentName;
        
        for (int j = 0; j < numAttrib; j++) {
            float currentAttrib;
            myfile >> currentAttrib;
            (*unknowns)[i*numAttrib + j] = currentAttrib;
        }
    }
    
    myfile.close();
}



//this is pseudocode
int main() {
    unsigned int numBlocks = 512;
    unsigned int threadsPerBlock = 256;
    
    // Metadata about our learning algorithm data
    int numAttributes, numKnownSamples, numClass, numUnknowns;
    
    // Data that needs to be sent to the device.
    float *h_min, *h_max;
    float *h_knowns;
    int *h_classifications;
    float *h_unknowns;
    
    // Device data
    float *d_min, *d_max;
    float *d_knowns;
    int *d_classifications;
    float *d_unknowns;
    
    string *unknownNames;
    
    // Needed for the profiling
    std::clock_t start;
    float duration;
    
    parse(&numAttributes, &numKnownSamples, &numClass, &numUnknowns, 
        &h_min, &h_max, &h_knowns, &h_classifications, &h_unknowns, &unknownNames);
    
    // Start mallocing the data to the kernel
    hipMalloc(&d_min, sizeof(float) * numAttributes);
    hipMalloc(&d_max, sizeof(float) * numAttributes);
    hipMalloc(&d_knowns, sizeof(float) * numKnownSamples * numAttributes);
    hipMalloc(&d_unknowns, sizeof(float) * numUnknowns * numAttributes);
    hipMalloc(&d_classifications, sizeof(int) * numKnownSamples);
    
    // Copy the data from the host to the kernel
    hipMemcpy(d_min, h_min, sizeof(float) * numAttributes, hipMemcpyHostToDevice);
    hipMemcpy(d_max, h_max, sizeof(float) * numAttributes, hipMemcpyHostToDevice);
    hipMemcpy(d_knowns, h_knowns, sizeof(float) * numKnownSamples * numAttributes, hipMemcpyHostToDevice);
    hipMemcpy(d_unknowns, h_unknowns, sizeof(float) * numUnknowns * numAttributes, hipMemcpyHostToDevice);
    hipMemcpy(d_classifications, h_classifications, sizeof(int) * numKnownSamples, hipMemcpyHostToDevice);

    
    // Normalize the known values
    threadsPerBlock = 256;
    numBlocks = numAttributes * numKnownSamples / threadsPerBlock + 1;
    normalize<<<numBlocks, threadsPerBlock>>>(d_knowns, d_max, d_min, 
        numAttributes, numKnownSamples);
    
    // Normalize the unknown values
    threadsPerBlock = 256;
    numBlocks = numAttributes * numKnownSamples / threadsPerBlock + 1;
    normalize<<<numBlocks, threadsPerBlock>>>(d_unknowns, d_max, d_min, 
        numAttributes, numUnknowns);
        
    
    // Generate the 
    float *d_distance;
    hipMalloc(&d_distance, sizeof(float) * numKnownSamples);
    threadsPerBlock = 256;
    numBlocks = numAttributes / threadsPerBlock + 1;
    
    findDistance<<<numBlocks, threadsPerBlock>>>(d_knowns, d_unknowns+0,  d_distance, 
        numAttributes, numKnownSamples);
    
    float *h_distance = (float*) malloc(sizeof(float) * numKnownSamples);
    hipMemcpy(h_distance, d_distance, sizeof(float) * numKnownSamples, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < numKnownSamples; i++) {
        printf("%f ", h_distance[i]); 
    }
    printf("\n");
    
    /*cudaMemcpy(h_unknowns, d_unknowns, sizeof(float) * numUnknowns * numAttributes, cudaMemcpyDeviceToHost);
    
    for (int i = 0; i < 5; i++) {
        printf("%f ", h_unknowns[i]); 
    }
    printf("\n");*/

    //float * distance;
    //cudaMalloc(&distance, sizeof(float) * numSomething
    //findDistance<<<numBlocks, threadsPerBlock>>>(d_knowns, d_unknowns, d_distance,  
    
    hipMemcpy(h_unknowns, d_unknowns, sizeof(float) * numUnknowns * numAttributes, hipMemcpyDeviceToHost);
}
